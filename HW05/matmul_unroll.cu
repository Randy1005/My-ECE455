#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matmul_unrolled(float* A, float* B_T, float* C, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int row = tid / N;
    int col = tid % N;
    float sum = 0;
    if (row < N && col < N) {
        for (int k = 0; k < N; k += 4) {
            sum += A[row * N + k + 0] * B_T[col * N + k + 0];
            sum += A[row * N + k + 1] * B_T[col * N + k + 1];
            sum += A[row * N + k + 2] * B_T[col * N + k + 2];
            sum += A[row * N + k + 3] * B_T[col * N + k + 3];
        }
        C[row * N + col] = sum;
    }
}

int main() {
    const int N = 1024;
    size_t size = N * N * sizeof(float);
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_B_T = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    for (int i = 0; i < N; ++i)
        for (int j = 0; j < N; ++j) {
            h_A[i * N + j] = 1.0f;
            h_B[i * N + j] = 1.0f;
            h_B_T[j * N + i] = h_B[i * N + j];
        }

    float *d_A, *d_B_T, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B_T, size);
    hipMalloc(&d_C, size);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B_T, h_B_T, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N * N + threadsPerBlock - 1) / threadsPerBlock;
    matmul_unrolled<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B_T, d_C, N);
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("C[0][0] = %f\n", h_C[0]);

    hipFree(d_A);
    hipFree(d_B_T);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_B_T);
    free(h_C);
    return 0;
}

